#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "CUDACore/AtomicPairCounter.h"
#include "CUDACore/cuteCheck.h"
#include "CUDACore/cuda_assert.h"

__global__ void update(cms::cuda::AtomicPairCounter *dc, uint32_t *ind, uint32_t *cont, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  auto m = i % 11;
  m = m % 6 + 1;  // max 6, no 0
  auto c = dc->add(m);
  assert(c.m < n);
  ind[c.m] = c.n;
  for (int j = c.n; j < c.n + m; ++j)
    cont[j] = i;
};

__global__ void finalize(cms::cuda::AtomicPairCounter const *dc, uint32_t *ind, uint32_t *cont, uint32_t n) {
  assert(dc->get().m == n);
  ind[n] = dc->get().n;
}

__global__ void verify(cms::cuda::AtomicPairCounter const *dc, uint32_t const *ind, uint32_t const *cont, uint32_t n) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  assert(0 == ind[0]);
  assert(dc->get().m == n);
  assert(ind[n] == dc->get().n);
  auto ib = ind[i];
  auto ie = ind[i + 1];
  auto k = cont[ib++];
  assert(k < n);
  for (; ib < ie; ++ib)
    assert(cont[ib] == k);
}

int main() {
  cms::cuda::AtomicPairCounter *dc_d;
  cuteCheck(hipMalloc((hipDeviceptr_t *)&dc_d, sizeof(cms::cuda::AtomicPairCounter)));
  cuteCheck(hipMemsetD8((hipDeviceptr_t) dc_d, 0, sizeof(cms::cuda::AtomicPairCounter)));

  std::cout << "size " << sizeof(cms::cuda::AtomicPairCounter) << std::endl;

  constexpr uint32_t N = 20000;
  constexpr uint32_t M = N * 6;
  uint32_t *n_d, *m_d;
  cuteCheck(hipMalloc((hipDeviceptr_t *)&n_d, N * sizeof(int)));
  // cuteCheck(hipMemsetD8((hipDeviceptr_t) n_d, 0, N*sizeof(int)));
  cuteCheck(hipMalloc((hipDeviceptr_t *)&m_d, M * sizeof(int)));

  update<<<2000, 512>>>(dc_d, n_d, m_d, 10000);
  finalize<<<1, 1>>>(dc_d, n_d, m_d, 10000);
  verify<<<2000, 512>>>(dc_d, n_d, m_d, 10000);

  cms::cuda::AtomicPairCounter dc;
  cuteCheck(cuMemcpy((hipDeviceptr_t)&dc, (hipDeviceptr_t)dc_d, sizeof(cms::cuda::AtomicPairCounter)));

  std::cout << dc.get().n << ' ' << dc.get().m << std::endl;

  return 0;
}
